#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

typedef unsigned long long bignum;

__host__ __device__ int isPrime(bignum x)
{

   bignum i;
   bignum lim = (bignum) sqrt((float)x) + 1.0;
      
   for(i=2; i<lim; i++){
      if ( x % i == 0)
         return 0;
   }//end of for loop

   return 1;
}//end of isPrime

__global__ void findPrime(int * arr, bignum n)
{
	bignum id = blockIdx.x * blockDim.x + threadIdx.x;
	bignum num = (2*(id + 1) -1);
	if(num < n)
	{
		arr[num] = isPrime(num);
		//arr[num] = 10;
	}//end of if	
}//end of kernal



int main(int argc, char ** argv)
{
	clock_t start = clock();
	if(argc < 2)
	{
		perror("Not Enough Arguments");
		exit(-1);
	}//end of if
	bignum N = atoi(argv[1]);
	int blockSize = atoi(argv[2]);
	bignum gridsize = (bignum)ceil((N+1)/2.0/blockSize);

	if(N <= 0)
	{
		printf("N isn't a valid number");
		exit(-99);
	}//end of if statment
	int * d_array;
	
	hipMalloc(&d_array, sizeof(int)*(N+1));
	hipMemset(d_array, 0, sizeof(int)*(N+1));
	
	findPrime<<<gridsize, blockSize>>>(d_array, N+1);
	int * h_array = (int *)calloc(N+1, sizeof(int));
	hipMemcpy(h_array, d_array, (N+1)*sizeof(int), hipMemcpyDeviceToHost);
	int i;
	int prime =0;
	for(i = 0; i <= N; i++)
	{
		printf("%d is %d\n", i, h_array[i]);
		if(h_array[i] == 1)
			prime++;
	}//end for loop
	free(h_array);
	hipFree(d_array);
	
	
	clock_t end = clock();
	printf("primes = %d, time = %f\n", prime,(float) (end-start)/ CLOCKS_PER_SEC);
	
}//end of main
