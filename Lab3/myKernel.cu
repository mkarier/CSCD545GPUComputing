#include "hip/hip_runtime.h"
#include "myKernel.h"

__global__ void kernel( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;

    a[idx]  = a[idx]+1;
}

// Please implement the following kernels2 through kernel6,
// in order to meet the requirements in the write-ups. 
__global__ void kernel2( int *a, int dimx, int dimy )
{
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;
	int idx = iy * dimx + ix;
	if(iy < dimy && ix < dimx)
		a[idx] = (blockIdx.y * gridDim.x) + blockIdx.x;
}

__global__ void kernel3( int *a, int dimx, int dimy )
{
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;
	int idx = iy*dimx + ix;
	if(iy < dimy && ix < dimx)
		a[idx] = idx;
}

__global__ void kernel4( int *a, int dimx, int dimy )
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = iy * dimx + ix;
	if(ix<dimx && iy < dimy)
		a[idx] = (threadIdx.y *  blockDim.x) + threadIdx.x;
}

__global__ void kernel5( int *a, int dimx, int dimy )
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = iy * dimx + ix;
	if(ix<dimx && iy < dimy)
		a[idx] = blockIdx.y;
}

__global__ void kernel6( int *a, int dimx, int dimy )
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = iy*dimx + ix;
	if(ix < dimx && iy < dimy)
		a[idx] = blockIdx.x;
}


