
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCKSIZE 4 // Number of threads in each thread block
 
// CUDA kernel. Each thread takes care of one element of a 
__global__ void diffKernel( float *in, float *out, int n )
{
    // Wrtie the kernel to implement the diff operation on an array 
	int id = (blockDim.x * blockIdx.x) + threadIdx.x;
	if(id < n-1)
		out[id] = in[id+1] - in[id];

}  
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int i;
    float input[] = {4, 5, 6, 7, 19, 10, 0, 4, 2, 3, 1, 7, 9, 11, 45, 23, 99, 29};
    int n = sizeof(input) / sizeof(float); //careful, this usage only works with statically allocated arrays, NOT dynamic arrays

    // Host input vectors
    float *h_in = input;
    //Host output vector
    float *h_out = (float *) malloc((n - 1) * sizeof(float));
 
    // Device input vectors
    float *d_in;;
    //Device output vector
    float *d_out;
 
    // Size, in bytes, of each vector
    size_t bytes = n * sizeof(float);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, bytes - sizeof(float));
 
    // Copy host data to device
    hipMemcpy( d_in, h_in, bytes, hipMemcpyHostToDevice);

    // TODO: setup the blocksize and gridsize and launch the kernel below.
    // Number of threads in each thread block
	int gridSize = (int)ceil((float)n/BLOCKSIZE);
 
    // Number of thread blocks in grid
 	int blockSize = BLOCKSIZE;
    // Execute the kernel
	diffKernel<<< gridSize, blockSize>>>(d_in, d_out, n);
    // Copy array back to host
    hipMemcpy( h_out, d_out, bytes - sizeof(float), hipMemcpyDeviceToHost );
 
    // Show the result
    printf("The original array is: ");
    for(i = 0; i < n; i ++)
        printf("%4.0f,", h_in[i] );    
    
    printf("\n\nThe diff     array is: ");
    for(i = 0; i < n - 1; i++)
        printf("%4.0f,", h_out[i] );    
    puts("");
    
    // Release device memory
    hipFree(d_in);
    hipFree(d_out);
 
    // Release host memory
    free(h_out);
 
    return 0;
}
