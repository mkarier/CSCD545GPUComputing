
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#define R 4
#define C 40 

/*
 * It returns the length of a string pointed by pointer s,
 * It acts like the cpu strlen() function
 */
__device__ int gpu_strlen(char * s)
{
	int size = 0;
	while(s[size] != '\n' && s[size] != '\0')
	{
		size++;
	}//end of while loop
	return size;
}//end of gpu_strlen

/*
 * It returns 0 if input character ch is NOT an alphabetical letter
 * Otherwise, it returns one.
 */
__device__ int gpu_isAlpha(char ch)
{
	char lowerletter = 'a', upperletter = 'A';
	int i = 0;
	for(i = 0; i < 26; i++, lowerletter++, upperletter++)
	{
		if(lowerletter == ch || upperletter == ch)
			return 1;
		if(ch == ' ' || ch == '\0')
			return 0;
	}
	return 0;
}

/* Cuda kernel to count number of words in each line of text pointed by a.
 * The output is stored back in 'out' array.
 * numLine specifies the num of lines in a, maxLineLen specifies the maximal
 * num of characters in one line of text.
 */
__global__ void wordCount( char **a, int **out, int numLine, int maxLineLen )
{
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	if(col < maxLineLen && row < numLine)
	{
		out[row][col] = 0;
		if(col < gpu_strlen(a[row]))
		{
			if(gpu_isAlpha(a[row][col]))
				out[row][col] = 0;
			else
				out[row][col] = 1;
		}//end of inner if
	}//end of if statement to check the wordCount

}//end of wordCount

/* Print out the all lines of text in a on stdout
 */ 
void printArr( char **a, int lines )
{
    int i;
    for(i=0; i<lines; i++)
    {
        printf("%s\n", a[i]);
    }
}


int main()
{
    int i; 
    char **d_in, **h_in, **h_out;
    int h_count_in[R][C], **h_count_out, **d_count_in;

    //allocate
    h_in = (char **)malloc(R * sizeof(char *));
    h_out = (char **)malloc(R * sizeof(char *));
    h_count_out = (int **)malloc(R * sizeof(int *));

    hipMalloc((void ***)&d_in, sizeof(char *) * R);
    hipMalloc((void ***)&d_count_in, sizeof(int *) * R);

    //alocate for string data
    for(i = 0; i < R; ++i) 
    {
        hipMalloc((void **) &h_out[i],C * sizeof(char));
        h_in[i]=(char *)calloc(C, sizeof(char));//allocate or connect the input data to it
        strcpy(h_in[i], "good morning and I'm a good student!");
        hipMemcpy(h_out[i], h_in[i], strlen(h_in[i]) + 1, hipMemcpyHostToDevice);
    }
    hipMemcpy(d_in, h_out, sizeof(char *) * R,hipMemcpyHostToDevice);

    //alocate for output occurrence
    for(i = 0; i < R; ++i) 
    {
        hipMalloc((void **) &h_count_out[i], C * sizeof(int));
        hipMemset(h_count_out[i], 0, C * sizeof(int));
    }
    hipMemcpy(d_count_in, h_count_out, sizeof(int *) * R,hipMemcpyHostToDevice);

    printArr(h_in, R);
    printf("\n\n");
     
    //set up kernel configuartion variables
    dim3 grid, block;
    block.x = 2;
    block.y = 2;
    grid.x  = ceil((float)C / block.x);
    grid.y  = ceil((float)R / block.y); //careful must be type cast into float, otherwise, integer division used
    printf("grid.x = %d, grid.y=%d\n", grid.x, grid.y );

    //launch kernel
    wordCount<<<grid, block>>>( d_in, d_count_in, R, C);

    //copy data back from device to host
    for(i = 0; i < R; ++i) {
        hipMemcpy(h_count_in[i], h_count_out[i], sizeof(int) * C,hipMemcpyDeviceToHost);
    }
    printf("Occurrence array obtained from device:\n");

    for(i = 0; i < R; i ++) {
        for(int j = 0; j < C; j ++)
            printf("%4d", h_count_in[i][j]);
        printf("\n");
    }
 
    return 0;
}

