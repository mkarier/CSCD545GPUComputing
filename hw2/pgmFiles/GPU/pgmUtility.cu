#include "hip/hip_runtime.h"

#include "Shared.h"
#include "GPU_PgmUtility.h"

// TODO:Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: You can NOT change the input, output, and argument type of the functions in pgmUtility.h
// NOTE: You can NOT change the prototype of any functions listed in pgmUtility.h
__device__ float distance( int p1[], int p2[] )
{
	return sqrtf(powf((p1[0] - p1[1]),2) + powf((p2[0] - p2[1]), 2));
}

/**
 *  Function Name:
 *      pgmRead()
 *      pgmRead() reads in a pgm image using file I/O, you have to follow the file format. All code in this function are exectured on CPU.
 *
 *  @param[in,out]  header  holds the header of the pgm file in a 2D character array
 *                          After we process the pixels in the input image, we write the origianl
 *                          header (or potentially modified) back to a new image file.
 *  @param[in,out]  numRows describes how many rows of pixels in the image.
 *  @param[in,out]  numCols describe how many pixels in one row in the image.
 *  @param[in]      in      FILE pointer, points to an opened image file that we like to read in.
 *  @return         If successful, return all pixels in the pgm image, which is an int **, equivalent to
 *                  a 2D array. Otherwise null.
 *
 */
__host__ int* pgmRead( char **header, int *numRows, int *numCols, FILE *in  )
{
	//TODO:
	char buffer[maxSizeHeadRow];
	int i = 0, j = 0;
	for( ; i < rowsInHeader; i++)
	{
		fgets(buffer, maxSizeHeadRow, in);
		header[i] =(char *) calloc((strlen(buffer)+1), sizeof(char));
		memcpy(header[i], buffer, strlen(buffer));
	}//end of for loop
	sscanf(header[2], "%d %d", numCols, numRows);
	int * array = (int *)malloc((*numRows) * (*numCols) * sizeof(int));

	for(i = 0; i < *numRows; i++)
	{
		for(j = 0; j < *numCols; j++)
		{
			fscanf(in, "%d", &array[i * *numCols + j]);
		}//end of inner for loop
	}//end of outer for loop

	return array;
}//end of pgmRead()


/**
 *  Function Name:
 *      pgmDrawCircle()
 *      pgmDrawCircle() draw a circle on the image by setting relavant pixels to Zero.
 *                      In this function, you have to invoke a CUDA kernel to perform all image processing on GPU.
 *
 *  @param[in,out]  pixels  holds all pixels in the pgm image, which a 2D integer array. The array
 *                          are modified after the drawing.
 *  @param[in]      numRows describes how many rows of pixels in the image.
 *  @param[in]      numCols describes how many columns of pixels in one row in the image.
 *  @param[in]      centerCol specifies at which column you like to center your circle.
 *  @param[in]      centerRow specifies at which row you like to center your circle.
 *                        centerCol and centerRow defines the center of the circle.
 *  @param[in]      radius    specifies what the radius of the circle would be, in number of pixels.
 *  @param[in,out]  header returns the new header after draw.
 *                  the circle draw might change the maximum intensity value in the image, so we
 *                  have to change maximum intensity value in the header accordingly.
 *  @return         return 1 if max intensity is changed, otherwise return 0;
 */
__global__ void pgmDrawCircle( int * pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header )
{
	//TODO:
	int id = (blockDim.x *blockIdx.x) + threadIdx.x;
	int p1[2];
	int p2[2];
	int y =  id % numCols;
	int x = id / numCols;
	p1[0] = y;
	p2[0] = x;
	p1[1] = centerRow;
	p2[1] = centerCol;
	if(id < (numRows * numCols))
	{
		float d = distance(p1, p2);//sqrtf(powf((p1[0] - p1[1]),2) + powf((p2[0] - p2[1]), 2));
		if(d < radius)
		{
			pixels[id] = 0;
		}
	}//end of if statement to check that id is in a grid
}//end of pgmDrawCircle.


/**
 *  Function Name:
 *      pgmDrawEdge()
 *      pgmDrawEdge() draws a black edge frame around the image by setting relavant pixels to Zero.
 *                    In this function, you have to invoke a CUDA kernel to perform all image processing on GPU.
 *
 *  @param[in,out]  pixels  holds all pixels in the pgm image, which a 2D integer array. The array
 *                          are modified after the drawing.
 *  @param[in]      numRows describes how many rows of pixels in the image.
 *  @param[in]      numCols describes how many columns of pixels in one row in the image.
 *  @param[in]      edgeWidth specifies how wide the edge frame would be, in number of pixels.
 *  @param[in,out]  header returns the new header after draw.
 *                  the function might change the maximum intensity value in the image, so we
 *                  have to change the maximum intensity value in the header accordingly.
 *
 *  @return         return 1 if max intensity is changed by the drawing, otherwise return 0;
 */
__global__ void pgmDrawEdge( int * pixels, int numRows, int numCols, int edgeWidth, char **header )
{
	int id = (blockDim.x * blockIdx.x) + threadIdx.x;
	int y =  id % numCols;
	int x = id / numCols;
	if(id < (numRows* numCols))
	{
		if(x < edgeWidth || x > (numRows - edgeWidth -1))
		{
			pixels[id] = 0;
		}//end of if satement to check the top and bottom of the picture.
		if(y < edgeWidth || y > (numCols -edgeWidth - 1))
		{
			pixels[id] = 0;
		}
	}//end of if statment to make sure that it is in thread is in the gird
}//end of pgmDrawEdge


/**
 *  Function Name:
 *      pgmDrawLine()
 *      pgmDrawLine() draws a straight line in the image by setting relavant pixels to Zero.
 *                      In this function, you have to invoke a CUDA kernel to perform all image processing on GPU.
 *
 *  @param[in,out]  pixels  holds all pixels in the pgm image, which a 2D integer array. The array
 *                          are modified after the drawing.
 *  @param[in]      numRows describes how many rows of pixels in the image.
 *  @param[in]      numCols describes how many columns of pixels in one row in the image.
 *  @param[in]      p1row specifies the row number of the start point of the line segment.
 *  @param[in]      p1col specifies the column number of the start point of the line segment.
 *  @param[in]      p2row specifies the row number of the end point of the line segment.
 *  @param[in]      p2col specifies the column number of the end point of the line segment.
 *  @param[in,out]  header returns the new header after draw.
 *                  the function might change the maximum intensity value in the image, so we
 *                  have to change the maximum intensity value in the header accordingly.
 *
 *  @return         return 1 if max intensity is changed by the drawing, otherwise return 0;
 */
__global__ void pgmDrawLine( int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col )
{
	double slope = (p2row - p1row)/(p2col - p1col);
	double b = p2row - (p2col * slope);
	int id = (blockDim.x * blockIdx.x) + threadIdx.x;
	int y =  id / numCols;
	int x = id % numCols;
	if(id < (numRows * numCols))
	{
		if((x >= p1col && x <= p2col) && (y >= p1row && y <= p2row))
		{
			if(y == ((slope * x) + b))
				pixels[id] = 0;
		}//end of if statement
	}//end of outer if statement
}//end of pgmDrawLine



/**
 *  Function Name:
 *      pgmWrite()
 *      pgmWrite() writes headers and pixels into a pgm image using file I/O.
 *                 writing back image has to strictly follow the image format. All code in this function are exectured on CPU.
 *
 *  @param[in]  header  holds the header of the pgm file in a 2D character array
 *                          we write the header back to a new image file on disk.
 *  @param[in]  pixels  holds all pixels in the pgm image, which a 2D integer array.
 *  @param[in]  numRows describes how many rows of pixels in the image.
 *  @param[in]  numCols describe how many columns of pixels in one row in the image.
 *  @param[in]  out     FILE pointer, points to an opened text file that we like to write into.
 *  @return     return 0 if the function successfully writes the header and pixels into file.
 *                          else return -1;
 */
__host__ int pgmWrite( const char **header, const int *pixels, int numRows, int numCols, FILE *out )
{
	//printf("It is in the pgmWrite method\n");
	int i, j;
	//printf("numRows, numCols, %d, %d\n", numRows, numCols);
	for(i = 0; i < rowsInHeader; i++)
	{
	  //printf("%s", header[i]);
	  fprintf(out, "%s", header[i]);
	}
	printf("finished printing the header\n");
	for(i = 0; i < numRows; i++)
	{
	  for(j = 0; j < numCols; j++)
	  {
		 //printf("%5d", pixels[i][j]);
		 fprintf(out, "%d ", pixels[i * numCols + j]);
	  }//end of loop for numCols
	  //printf("\n");
	  fprintf(out,"\n");
	}//end of for loop for numRows
	return 0;
}//end of pgmWrite

