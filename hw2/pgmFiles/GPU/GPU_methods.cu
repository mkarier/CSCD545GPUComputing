#include "hip/hip_runtime.h"
/*
 * Author; Marco Karier
 *
 * This is where the main method for the cuda
 */

#include "Shared.h"
#include "GPU_PgmUtility.h"
#define BLOCKSIZE 16

void callCircle(int centerRow, int centerColumn, int radius, FILE * in, FILE * out)
{
	char ** header = (char **)calloc(rowsInHeader, sizeof(char *));
	int numRows= 0, numCols = 0;
	printf("Reading in file for single array\n");
	int * h_pixels = pgmRead(header, &numRows, &numCols, in);
	int size = numRows * numCols *sizeof(int);
	printf("read in file\n");
	fclose(in);

	int * d_pixels;
	hipMalloc(&d_pixels, size);
	hipMemset(d_pixels, 0, size);
	hipMemcpy(d_pixels, h_pixels, size, hipMemcpyHostToDevice);
	int gridSize =(int)ceil((double)size/ BLOCKSIZE);// (int)ceil((float)n/BLOCKSIZE);
	pgmDrawCircle<<<gridSize, BLOCKSIZE>>>(d_pixels, numRows, numCols, centerRow, centerColumn, radius, header);
	hipMemcpy(h_pixels, d_pixels, size, hipMemcpyDeviceToHost);

	pgmWrite((const char **)header, (const int *)h_pixels, numRows, numCols, out);
	int i;
	for(i = 0; i < rowsInHeader; i++)
	{
		free(header[i]);
	}//end of for loop to clear header
	free(header);
	free(h_pixels);
	hipFree(d_pixels);
	fclose(out);
}//end of callCircle

void callEdge(int edgeWidth, FILE * in, FILE * out)
{
	char ** header = (char **)calloc(rowsInHeader, sizeof(char *));
	int numRows= 0, numCols = 0;
	printf("Reading in file for single array\n");
	int * h_pixels = pgmRead(header, &numRows, &numCols, in);
	int size = numRows * numCols *sizeof(int);
	printf("read in file\n");
	fclose(in);

	int * d_pixels;
	hipMalloc(&d_pixels, size);
	hipMemset(d_pixels, 0, size);
	hipMemcpy(d_pixels, h_pixels, size, hipMemcpyHostToDevice);
	int gridSize =(int)ceil((double)size/ BLOCKSIZE);// (int)ceil((float)n/BLOCKSIZE);
	pgmDrawEdge<<<gridSize, BLOCKSIZE>>>(d_pixels, numRows, numCols, edgeWidth, header);
	hipMemcpy(h_pixels, d_pixels, size, hipMemcpyDeviceToHost);

	pgmWrite((const char **)header, (const int *)h_pixels, numRows, numCols, out);
	int i;
	for(i = 0; i < rowsInHeader; i++)
	{
		free(header[i]);
	}//end of for loop to clear header
	free(header);
	free(h_pixels);
	hipFree(d_pixels);
	fclose(out);
}//end of callEdge

void callLine(int p1row, int p1col, int p2row, int p2col, FILE * in,  FILE * out)
{
	char ** header = (char **)calloc(rowsInHeader, sizeof(char *));
	int numRows= 0, numCols = 0;
	printf("Reading in file for single array\n");
	int * h_pixels = pgmRead(header, &numRows, &numCols, in);
	int size = numRows * numCols *sizeof(int);
	printf("read in file\n");
	fclose(in);

	int * d_pixels;
	hipMalloc(&d_pixels, size);
	hipMemset(d_pixels, 0, size);
	hipMemcpy(d_pixels, h_pixels, size, hipMemcpyHostToDevice);
	int gridSize =(int)ceil((double)size/ BLOCKSIZE);// (int)ceil((float)n/BLOCKSIZE);
	pgmDrawLine<<<gridSize, BLOCKSIZE>>>(d_pixels, numRows, numCols, header, p1row, p1col, p2row, p2col);
	hipMemcpy(h_pixels, d_pixels, size, hipMemcpyDeviceToHost);

	pgmWrite((const char **)header, (const int *)h_pixels, numRows, numCols, out);
	int i;
	for(i = 0; i < rowsInHeader; i++)
	{
		free(header[i]);
	}//end of for loop to clear header
	free(header);
	free(h_pixels);
	hipFree(d_pixels);
	fclose(out);
}//end of calline

