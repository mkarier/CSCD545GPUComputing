
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#define R 5
#define C 40 

__device__ int gpu_strlen(char * s)
{
    int i = 0;
    while(s[i++] != '\0')
    {
    }
    return i;
}

__device__ int gpu_isAlpha(char ch)
{
    if((ch >= 'a' && ch <= 'z') || (ch >= 'A' && ch <= 'Z'))
        return 1;
    else
        return 0;
}

__global__ void wordCount2( char **a, int **out, int numLine, int maxLineLen )
{
	extern __shared__ char s_data[];
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockIdx.y;
	if(col < maxLineLen && row < numLine)
	{
		s_data[col] = a[row][col];
	}
	__syncthreads();
	if(col < maxLineLen && row < numLine && col < gpu_strlen(a[row]))
	{
		if(gpu_isAlpha(s_data[col]))
			out[row][col] = 0;
		else
		{
			if(col != 0 && gpu_isAlpha(s_data[col-1]))
				out[row][col] = 1;
			else
				out[row][col] = 0;
		}
	}


}//end of wordCount2

void checkErr()
{
    hipError_t code = hipGetLastError();
    if (code != hipSuccess)
        printf ("Cuda error -- %s\n", hipGetErrorString(code));
}

void printArr( char **a, int lines )
{
    int i;
    for(i=0; i<lines; i++)
    {
        printf("%s\n", a[i]);
    }
}


int main()
{
    int i, j; 
    char **d_in, **h_in, **h_out;
    int h_count_in[R][C], **h_count_out, **d_count_in;

    for(i = 0; i < R; i++)
        for(j = 0; j < C; j ++)
            h_count_in[i][j] = 0;

    //allocate
    h_in = (char **)malloc(R * sizeof(char *));
    h_out = (char **)malloc(R * sizeof(char *));
    h_count_out = (int **)malloc(R * sizeof(int *));
  
    hipDeviceReset();

    hipMalloc((void ***)&d_in, sizeof(char *) * R);
    hipMalloc((void ***)&d_count_in, sizeof(int *) * R);

    //alocate for string data
    for(i = 0; i < R; ++i) 
    {
        hipMalloc((void **) &h_out[i],C * sizeof(char));
        h_in[i]=(char *)calloc(C, sizeof(char));//allocate or connect the input data to it
        strcpy(h_in[i], "good morning and I'm a good student!");
        hipMemcpy(h_out[i], h_in[i], strlen(h_in[i]) + 1, hipMemcpyHostToDevice);
    }
    hipMemcpy(d_in, h_out, sizeof(char *) * R,hipMemcpyHostToDevice);

    //alocate for output occurrence
    for(i = 0; i < R; ++i) 
    {
        hipMalloc((void **) &h_count_out[i], C * sizeof(int));
        hipMemset(h_count_out[i], 0, C * sizeof(int));
    }
    hipMemcpy(d_count_in, h_count_out, sizeof(int *) * R,hipMemcpyHostToDevice);

    printArr(h_in, R);
    printf("\n\n");
     
    //set up kernel configuration variables
    dim3 grid, block;
    block.x = C;   //NOTE: differs from last lab6 here, Why?
    block.y = 1;
    grid.x  = ceil((float)C / block.x);
    grid.y  = ceil((float)R / block.y); //careful must be type cast into float, otherwise, integer division used
    //printf("grid.x = %d, grid.y=%d\n", grid.x, grid.y );

    //launch kernel
    wordCount2<<<grid, block, grid.x>>>( d_in, d_count_in, R, C);
    checkErr();

    //copy data back from device to host
    for(i = 0; i < R; ++i) {
        hipMemcpy(h_count_in[i], h_count_out[i], sizeof(int) * C,hipMemcpyDeviceToHost);
    }

    printf("Occurrence array obtained from device:\n");

    for(i = 0; i < R; i ++) {
        for(int j = 0; j < C; j ++)
            printf("%4d", h_count_in[i][j]);
        printf("\n");
    }
 
    return 0;
}

